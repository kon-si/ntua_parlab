#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){
	return (blockIdx.x * blockDim.x + threadIdx.x); /* DONE: Calculate 1-Dim global ID of a thread */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
float euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numCoords][numObjs]
                    float *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    float ans=0.0;

	/* DONE: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
	for (i=0; i<numCoords; i++) {
    ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
           (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
    }

    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,           //  [numCoords][numObjs]
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,          //  [numObjs]
                          float *devdelta)
{

    /* Get the global ID of the thread. */
    int tid = get_tid();

    /* DONE: Maybe something is missing here... should all threads run this? */
    if (tid < numObjs) {
        int   index, i;
        float dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* DONE: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
        min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0);

        for (i=1; i<numClusters; i++) {
            /* DONE: call dist = euclid_dist_2(...) with correct objectId/clusterId */
            dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i);

            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[tid] != index) {
            /* DONE: Maybe something is missing here... is this write safe? */
            atomicAdd(devdelta, 1.0);
        }

        /* assign the membership to object objectId */
        membership[tid] = index;
    }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(	float *objects,      /* in: [numObjs][numCoords] */
		               	int     numCoords,    /* no. features */
		               	int     numObjs,      /* no. objects */
		               	int     numClusters,  /* no. clusters */
		               	float   threshold,    /* % objects change membership */
		               	long    loop_threshold,   /* maximum number of iterations */
		               	int    *membership,   /* out: [numObjs] */
						float * clusters,   /* out: [numClusters][numCoords] */
						int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0, gpu_cpu_timer = 0, gpu_timer = 0, cpu_timer = 0, cpu_gpu_timer = 0, start_timer; 
	int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    float  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    
    /* DONE: Transpose dims */
    float  **dimObjects = (float**) calloc_2d(numCoords, numObjs, sizeof(float));
    float  **dimClusters = (float**) calloc_2d(numCoords, numClusters, sizeof(float));
    float  **newClusters = (float**) calloc_2d(numCoords, numClusters, sizeof(float));
    
    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;

    printf("\n|-----------Transpose GPU Kmeans------------|\n\n");
    
    /* DONE: Copy objects given in [numObjs][numCoords] layout to new [numCoords][numObjs] layout */
	for(i = 0; i < numCoords; i++){
		for(j = 0; j < numObjs; j++){
			dimObjects[i][j] = objects[numCoords * j + i];
		} 
	}
	
    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL); 
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 

    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
	/* DONE: Calculate Grid size, e.g. number of blocks. */
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock -1) / numThreadsPerClusterBlock;
    const unsigned int clusterBlockSharedDataSize = 0;
       
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(float)));
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   
    
    do {
    	timing_internal = wtime();
        start_timer = timing_internal;

		/* GPU part: calculate new memberships */
		        
        /* DONE: Copy clusters to deviceClusters */
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));
        
		cpu_gpu_timer += wtime() - start_timer;

        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(float)));          

		start_timer = wtime();

		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
	
		gpu_timer += wtime() - start_timer;
        start_timer = wtime();
	
		/* DONE: Copy deviceMembership to membership */
        checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost));
    
    	/* DONE: Copy dev_delta_ptr to &delta */
        checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(float), hipMemcpyDeviceToHost));

        gpu_cpu_timer += wtime() - start_timer;

		/* CPU part: Update cluster centers*/
  		
		start_timer = wtime();

        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i*numCoords + j];
        }
 
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 

        cpu_timer += wtime() - start_timer;

        //printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
    
    /* DONE: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */ 
	for (i = 0; i < numClusters; i++) {
        for (j = 0; j < numCoords; j++) {
            clusters[numCoords * i + j] = dimClusters[j][i];
        }
    }
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/Sz-%ld_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(float)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 
	fprintf(fp, "%s,%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", "Transpose", blockSize, timing/loop, timer_min, timer_max, gpu_timer/loop, cpu_timer/loop, gpu_cpu_timer/loop, cpu_gpu_timer/loop);
	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}

